#include "hip/hip_runtime.h"
/*
 * =======================================================================================
 *
 *   Author:   Jan Eitzinger (je), jan.eitzinger@fau.de
 *   Copyright (c) 2020 RRZE, University Erlangen-Nuremberg
 *
 *   This file is part of MD-Bench.
 *
 *   MD-Bench is free software: you can redistribute it and/or modify it
 *   under the terms of the GNU Lesser General Public License as published
 *   by the Free Software Foundation, either version 3 of the License, or
 *   (at your option) any later version.
 *
 *   MD-Bench is distributed in the hope that it will be useful, but WITHOUT ANY
 *   WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A
 *   PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 *   details.
 *
 *   You should have received a copy of the GNU Lesser General Public License along
 *   with MD-Bench.  If not, see <https://www.gnu.org/licenses/>.
 * =======================================================================================
 */
#include <stdlib.h>
#include <stdio.h>

extern "C" {

#include <pbc.h>
#include <atom.h>
#include <allocate.h>

#define DELTA 20000

}

__global__ void computePbcUpdate(Atom a, int* PBCx, int* PBCy, int* PBCz, MD_FLOAT xprd, MD_FLOAT yprd, MD_FLOAT zprd){
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int Nghost = a.Nghost;
    if( i >= Nghost ) {
        return;
    }
    Atom* atom = &a;
    int *border_map = atom->border_map;
    int nlocal = atom->Nlocal;

    atom_x(nlocal + i) = atom_x(border_map[i]) + PBCx[i] * xprd;
    atom_y(nlocal + i) = atom_y(border_map[i]) + PBCy[i] * yprd;
    atom_z(nlocal + i) = atom_z(border_map[i]) + PBCz[i] * zprd;
}

extern "C"{

static int NmaxGhost;
static int *PBCx, *PBCy, *PBCz;

static int c_NmaxGhost = 0;
static int *c_PBCx = NULL, *c_PBCy = NULL, *c_PBCz = NULL;

static void growPbc(Atom *);

/* exported subroutines */
void initPbc(Atom *atom) {
    NmaxGhost = 0;
    atom->border_map = NULL;
    PBCx = NULL;
    PBCy = NULL;
    PBCz = NULL;
}

/* update coordinates of ghost atoms */
/* uses mapping created in setupPbc */
void updatePbc(Atom *atom, Parameter *param) {
    int *border_map = atom->border_map;
    int nlocal = atom->Nlocal;
    MD_FLOAT xprd = param->xprd;
    MD_FLOAT yprd = param->yprd;
    MD_FLOAT zprd = param->zprd;

    for (int i = 0; i < atom->Nghost; i++) {
        atom_x(nlocal + i) = atom_x(border_map[i]) + PBCx[i] * xprd;
        atom_y(nlocal + i) = atom_y(border_map[i]) + PBCy[i] * yprd;
        atom_z(nlocal + i) = atom_z(border_map[i]) + PBCz[i] * zprd;
    }
}

/* update coordinates of ghost atoms */
/* uses mapping created in setupPbc */
void updatePbc_cuda(Atom *atom, Parameter *param, Atom *c_atom, bool doReneighbor, const int num_threads_per_block) {
    if (doReneighbor){
        c_atom->Natoms = atom->Natoms;
        c_atom->Nlocal = atom->Nlocal;
        c_atom->Nghost = atom->Nghost;
        c_atom->ntypes = atom->ntypes;

        if (atom->Nmax > c_atom->Nmax){ // the number of ghost atoms has increased -> more space is needed
            c_atom->Nmax = atom->Nmax;
            if(c_atom->x != NULL){ hipFree(c_atom->x); }
            if(c_atom->type != NULL){ hipFree(c_atom->type); }
            checkCUDAError( "updatePbc c_atom->x malloc", hipMalloc((void**)&(c_atom->x), sizeof(MD_FLOAT) * atom->Nmax * 3) );
            checkCUDAError( "updatePbc c_atom->type malloc", hipMalloc((void**)&(c_atom->type), sizeof(int) * atom->Nmax) );
        }
        // TODO if the sort is reactivated the atom->vx needs to be copied to GPU as well
        checkCUDAError( "updatePbc c_atom->x memcpy", hipMemcpy(c_atom->x, atom->x, sizeof(MD_FLOAT) * atom->Nmax * 3, hipMemcpyHostToDevice) );
        checkCUDAError( "updatePbc c_atom->type memcpy", hipMemcpy(c_atom->type, atom->type, sizeof(int) * atom->Nmax, hipMemcpyHostToDevice) );

        if(c_NmaxGhost < NmaxGhost){
            c_NmaxGhost = NmaxGhost;
            if(c_PBCx != NULL){ hipFree(c_PBCx); }
            if(c_PBCy != NULL){ hipFree(c_PBCy); }
            if(c_PBCz != NULL){ hipFree(c_PBCz); }
            if(c_atom->border_map != NULL){ hipFree(c_atom->border_map); }
            checkCUDAError( "updatePbc c_PBCx malloc", hipMalloc((void**)&c_PBCx, NmaxGhost * sizeof(int)) );
            checkCUDAError( "updatePbc c_PBCy malloc", hipMalloc((void**)&c_PBCy, NmaxGhost * sizeof(int)) );
            checkCUDAError( "updatePbc c_PBCz malloc", hipMalloc((void**)&c_PBCz, NmaxGhost * sizeof(int)) );
            checkCUDAError( "updatePbc c_atom->border_map malloc", hipMalloc((void**)&(c_atom->border_map), NmaxGhost * sizeof(int)) );
        }
        checkCUDAError( "updatePbc c_PBCx memcpy", hipMemcpy(c_PBCx, PBCx, NmaxGhost * sizeof(int), hipMemcpyHostToDevice) );
        checkCUDAError( "updatePbc c_PBCy memcpy", hipMemcpy(c_PBCy, PBCy, NmaxGhost * sizeof(int), hipMemcpyHostToDevice) );
        checkCUDAError( "updatePbc c_PBCz memcpy", hipMemcpy(c_PBCz, PBCz, NmaxGhost * sizeof(int), hipMemcpyHostToDevice) );
        checkCUDAError( "updatePbc c_atom->border_map memcpy", hipMemcpy(c_atom->border_map, atom->border_map, NmaxGhost * sizeof(int), hipMemcpyHostToDevice) );
    }

    int nlocal = atom->Nlocal;
    MD_FLOAT xprd = param->xprd;
    MD_FLOAT yprd = param->yprd;
    MD_FLOAT zprd = param->zprd;

    const int num_blocks = ceil((float)atom->Nghost / (float)num_threads_per_block);

    /*__global__ void computePbcUpdate(Atom a, int* PBCx, int* PBCy, int* PBCz,
     *                                                          MD_FLOAT xprd, MD_FLOAT yprd, MD_FLOAT zprd)
     * */
    computePbcUpdate<<<num_blocks, num_threads_per_block>>>(*c_atom, c_PBCx, c_PBCy, c_PBCz, xprd, yprd, zprd);
    if(doReneighbor){
    	checkCUDAError( "updatePbc atom->x memcpy back", hipMemcpy(atom->x, c_atom->x, atom->Nmax * sizeof(MD_FLOAT) * 3, hipMemcpyDeviceToHost) );
    }
}

/* relocate atoms that have left domain according
 * to periodic boundary conditions */
void updateAtomsPbc(Atom *atom, Parameter *param) {
    MD_FLOAT xprd = param->xprd;
    MD_FLOAT yprd = param->yprd;
    MD_FLOAT zprd = param->zprd;

    for (int i = 0; i < atom->Nlocal; i++) {

        if (atom_x(i) < 0.0) {
            atom_x(i) += xprd;
        } else if (atom_x(i) >= xprd) {
            atom_x(i) -= xprd;
        }

        if (atom_y(i) < 0.0) {
            atom_y(i) += yprd;
        } else if (atom_y(i) >= yprd) {
            atom_y(i) -= yprd;
        }

        if (atom_z(i) < 0.0) {
            atom_z(i) += zprd;
        } else if (atom_z(i) >= zprd) {
            atom_z(i) -= zprd;
        }
    }
}

/* setup periodic boundary conditions by
 * defining ghost atoms around domain
 * only creates mapping and coordinate corrections
 * that are then enforced in updatePbc */
#define ADDGHOST(dx, dy, dz)                              \
    Nghost++;                                           \
    border_map[Nghost] = i;                             \
    PBCx[Nghost] = dx;                                  \
    PBCy[Nghost] = dy;                                  \
    PBCz[Nghost] = dz;                                  \
    atom->type[atom->Nlocal + Nghost] = atom->type[i]

void setupPbc(Atom *atom, Parameter *param) {
    int *border_map = atom->border_map;
    MD_FLOAT xprd = param->xprd;
    MD_FLOAT yprd = param->yprd;
    MD_FLOAT zprd = param->zprd;
    MD_FLOAT Cutneigh = param->cutneigh;
    int Nghost = -1;

    for (int i = 0; i < atom->Nlocal; i++) {

        if (atom->Nlocal + Nghost + 7 >= atom->Nmax) {
            growAtom(atom);
        }
        if (Nghost + 7 >= NmaxGhost) {
            growPbc(atom);
            border_map = atom->border_map;
        }

        MD_FLOAT x = atom_x(i);
        MD_FLOAT y = atom_y(i);
        MD_FLOAT z = atom_z(i);

        /* Setup ghost atoms */
        /* 6 planes */
        if (x < Cutneigh) { ADDGHOST(+1, 0, 0); }
        if (x >= (xprd - Cutneigh)) { ADDGHOST(-1, 0, 0); }
        if (y < Cutneigh) { ADDGHOST(0, +1, 0); }
        if (y >= (yprd - Cutneigh)) { ADDGHOST(0, -1, 0); }
        if (z < Cutneigh) { ADDGHOST(0, 0, +1); }
        if (z >= (zprd - Cutneigh)) { ADDGHOST(0, 0, -1); }
        /* 8 corners */
        if (x < Cutneigh && y < Cutneigh && z < Cutneigh) { ADDGHOST(+1, +1, +1); }
        if (x < Cutneigh && y >= (yprd - Cutneigh) && z < Cutneigh) { ADDGHOST(+1, -1, +1); }
        if (x < Cutneigh && y >= Cutneigh && z >= (zprd - Cutneigh)) { ADDGHOST(+1, +1, -1); }
        if (x < Cutneigh && y >= (yprd - Cutneigh) && z >= (zprd - Cutneigh)) { ADDGHOST(+1, -1, -1); }
        if (x >= (xprd - Cutneigh) && y < Cutneigh && z < Cutneigh) { ADDGHOST(-1, +1, +1); }
        if (x >= (xprd - Cutneigh) && y >= (yprd - Cutneigh) && z < Cutneigh) { ADDGHOST(-1, -1, +1); }
        if (x >= (xprd - Cutneigh) && y < Cutneigh && z >= (zprd - Cutneigh)) { ADDGHOST(-1, +1, -1); }
        if (x >= (xprd - Cutneigh) && y >= (yprd - Cutneigh) && z >= (zprd - Cutneigh)) { ADDGHOST(-1, -1, -1); }
        /* 12 edges */
        if (x < Cutneigh && z < Cutneigh) { ADDGHOST(+1, 0, +1); }
        if (x < Cutneigh && z >= (zprd - Cutneigh)) { ADDGHOST(+1, 0, -1); }
        if (x >= (xprd - Cutneigh) && z < Cutneigh) { ADDGHOST(-1, 0, +1); }
        if (x >= (xprd - Cutneigh) && z >= (zprd - Cutneigh)) { ADDGHOST(-1, 0, -1); }
        if (y < Cutneigh && z < Cutneigh) { ADDGHOST(0, +1, +1); }
        if (y < Cutneigh && z >= (zprd - Cutneigh)) { ADDGHOST(0, +1, -1); }
        if (y >= (yprd - Cutneigh) && z < Cutneigh) { ADDGHOST(0, -1, +1); }
        if (y >= (yprd - Cutneigh) && z >= (zprd - Cutneigh)) { ADDGHOST(0, -1, -1); }
        if (y < Cutneigh && x < Cutneigh) { ADDGHOST(+1, +1, 0); }
        if (y < Cutneigh && x >= (xprd - Cutneigh)) { ADDGHOST(-1, +1, 0); }
        if (y >= (yprd - Cutneigh) && x < Cutneigh) { ADDGHOST(+1, -1, 0); }
        if (y >= (yprd - Cutneigh) && x >= (xprd - Cutneigh)) { ADDGHOST(-1, -1, 0); }
    }
    // increase by one to make it the ghost atom count
    atom->Nghost = Nghost + 1;
}

/* internal subroutines */
void growPbc(Atom *atom) {
    int nold = NmaxGhost;
    NmaxGhost += DELTA;

    atom->border_map = (int *) reallocate(atom->border_map, ALIGNMENT, NmaxGhost * sizeof(int), nold * sizeof(int));
    PBCx = (int *) reallocate(PBCx, ALIGNMENT, NmaxGhost * sizeof(int), nold * sizeof(int));
    PBCy = (int *) reallocate(PBCy, ALIGNMENT, NmaxGhost * sizeof(int), nold * sizeof(int));
    PBCz = (int *) reallocate(PBCz, ALIGNMENT, NmaxGhost * sizeof(int), nold * sizeof(int));
}
}
