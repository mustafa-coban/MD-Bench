#include "hip/hip_runtime.h"
/*
 * =======================================================================================
 *
 *   Author:   Jan Eitzinger (je), jan.eitzinger@fau.de
 *   Copyright (c) 2021 RRZE, University Erlangen-Nuremberg
 *
 *   This file is part of MD-Bench.
 *
 *   MD-Bench is free software: you can redistribute it and/or modify it
 *   under the terms of the GNU Lesser General Public License as published
 *   by the Free Software Foundation, either version 3 of the License, or
 *   (at your option) any later version.
 *
 *   MD-Bench is distributed in the hope that it will be useful, but WITHOUT ANY
 *   WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A
 *   PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 *   details.
 *
 *   You should have received a copy of the GNU Lesser General Public License along
 *   with MD-Bench.  If not, see <https://www.gnu.org/licenses/>.
 * =======================================================================================
 */
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <>

extern "C" {

#include <neighbor.h>
#include <parameter.h>
#include <allocate.h>
#include <atom.h>

#define SMALL 1.0e-6
#define FACTOR 0.999
}

__device__ int coord2bin_device(MD_FLOAT xin, MD_FLOAT yin, MD_FLOAT zin, 
                                Neighbor_params np)
{
    int ix, iy, iz;

    if(xin >= np.xprd) {
        ix = (int)((xin - np.xprd) * np.bininvx) + np.nbinx - np.mbinxlo;
    } else if(xin >= 0.0) {
        ix = (int)(xin * np.bininvx) - np.mbinxlo;
    } else {
        ix = (int)(xin * np.bininvx) - np.mbinxlo - 1;
    }

    if(yin >= np.yprd) {
        iy = (int)((yin - np.yprd) * np.bininvy) + np.nbiny - np.mbinylo;
    } else if(yin >= 0.0) {
        iy = (int)(yin * np.bininvy) - np.mbinylo;
    } else {
        iy = (int)(yin * np.bininvy) - np.mbinylo - 1;
    }

    if(zin >= np.zprd) {
        iz = (int)((zin - np.zprd) * np.bininvz) + np.nbinz - np.mbinzlo;
    } else if(zin >= 0.0) {
        iz = (int)(zin * np.bininvz) - np.mbinzlo;
    } else {
        iz = (int)(zin * np.bininvz) - np.mbinzlo - 1;
    }

    return (iz * np.mbiny * np.mbinx + iy * np.mbinx + ix + 1);
}

__global__ void compute_neighborhood(Atom a, Neighbor neigh, Neighbor_params np, int nstencil, int* stencil,
                                     int* bins, int atoms_per_bin, int *bincount, int *new_maxneighs, MD_FLOAT cutneighsq){
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int Nlocal = a.Nlocal;
    if( i >= Nlocal ) {
        return;
    }
    
    Atom *atom = &a;
    Neighbor *neighbor = &neigh;
    
    int* neighptr = &(neighbor->neighbors[i]);
    int n = 0;
    MD_FLOAT xtmp = atom_x(i);
    MD_FLOAT ytmp = atom_y(i);
    MD_FLOAT ztmp = atom_z(i);
    int ibin = coord2bin_device(xtmp, ytmp, ztmp, np);
#ifdef EXPLICIT_TYPES
    int type_i = atom->type[i];
#endif
    for(int k = 0; k < nstencil; k++) {
        int jbin = ibin + stencil[k];
        int* loc_bin = &bins[jbin * atoms_per_bin];

        for(int m = 0; m < bincount[jbin]; m++) {
            int j = loc_bin[m];

            if ( j == i ){
                continue;
            }

            MD_FLOAT delx = xtmp - atom_x(j);
            MD_FLOAT dely = ytmp - atom_y(j);
            MD_FLOAT delz = ztmp - atom_z(j);
            MD_FLOAT rsq = delx * delx + dely * dely + delz * delz;

#ifdef EXPLICIT_TYPES
            int type_j = atom->type[j];
                    const MD_FLOAT cutoff = atom->cutneighsq[type_i * atom->ntypes + type_j];
#else
            const MD_FLOAT cutoff = cutneighsq;
#endif

            if( rsq <= cutoff ) {
                int idx = atom->Nlocal * n;
                neighptr[idx] = j;
                n += 1;
            }
        }
    }

    neighbor->numneigh[i] = n;

    if(n > neighbor->maxneighs) {
        atomicMax(new_maxneighs, n);
    }
}

extern "C" {
    
    
static MD_FLOAT xprd, yprd, zprd;
static MD_FLOAT bininvx, bininvy, bininvz;
static int mbinxlo, mbinylo, mbinzlo;
static int nbinx, nbiny, nbinz;
static int mbinx, mbiny, mbinz; // n bins in x, y, z
static int *bincount;
static int *bins;
static int mbins; //total number of bins
static int atoms_per_bin;  // max atoms per bin
static MD_FLOAT cutneigh;
static MD_FLOAT cutneighsq;  // neighbor cutoff squared
static int nmax;
static int nstencil;      // # of bins in stencil
static int* stencil;      // stencil list of bin offsets
static MD_FLOAT binsizex, binsizey, binsizez;

static int coord2bin(MD_FLOAT, MD_FLOAT , MD_FLOAT);
static MD_FLOAT bindist(int, int, int);

/* exported subroutines */
void initNeighbor(Neighbor *neighbor, Parameter *param)
{
    MD_FLOAT neighscale = 5.0 / 6.0;
    xprd = param->nx * param->lattice;
    yprd = param->ny * param->lattice;
    zprd = param->nz * param->lattice;
    cutneigh = param->cutneigh;
    nbinx = neighscale * param->nx;
    nbiny = neighscale * param->ny;
    nbinz = neighscale * param->nz;
    nmax = 0;
    atoms_per_bin = 8;
    stencil = NULL;
    bins = NULL;
    bincount = NULL;
    neighbor->maxneighs = 100;
    neighbor->numneigh = NULL;
    neighbor->neighbors = NULL;
}

void setupNeighbor()
{
    MD_FLOAT coord;
    int mbinxhi, mbinyhi, mbinzhi;
    int nextx, nexty, nextz;
    MD_FLOAT xlo = 0.0; MD_FLOAT xhi = xprd;
    MD_FLOAT ylo = 0.0; MD_FLOAT yhi = yprd;
    MD_FLOAT zlo = 0.0; MD_FLOAT zhi = zprd;

    cutneighsq = cutneigh * cutneigh;
    binsizex = xprd / nbinx;
    binsizey = yprd / nbiny;
    binsizez = zprd / nbinz;
    bininvx = 1.0 / binsizex;
    bininvy = 1.0 / binsizey;
    bininvz = 1.0 / binsizez;

    coord = xlo - cutneigh - SMALL * xprd;
    mbinxlo = (int) (coord * bininvx);
    if (coord < 0.0) {
        mbinxlo = mbinxlo - 1;
    }
    coord = xhi + cutneigh + SMALL * xprd;
    mbinxhi = (int) (coord * bininvx);

    coord = ylo - cutneigh - SMALL * yprd;
    mbinylo = (int) (coord * bininvy);
    if (coord < 0.0) {
        mbinylo = mbinylo - 1;
    }
    coord = yhi + cutneigh + SMALL * yprd;
    mbinyhi = (int) (coord * bininvy);

    coord = zlo - cutneigh - SMALL * zprd;
    mbinzlo = (int) (coord * bininvz);
    if (coord < 0.0) {
        mbinzlo = mbinzlo - 1;
    }
    coord = zhi + cutneigh + SMALL * zprd;
    mbinzhi = (int) (coord * bininvz);

    mbinxlo = mbinxlo - 1;
    mbinxhi = mbinxhi + 1;
    mbinx = mbinxhi - mbinxlo + 1;

    mbinylo = mbinylo - 1;
    mbinyhi = mbinyhi + 1;
    mbiny = mbinyhi - mbinylo + 1;

    mbinzlo = mbinzlo - 1;
    mbinzhi = mbinzhi + 1;
    mbinz = mbinzhi - mbinzlo + 1;

    nextx = (int) (cutneigh * bininvx);
    if(nextx * binsizex < FACTOR * cutneigh) nextx++;

    nexty = (int) (cutneigh * bininvy);
    if(nexty * binsizey < FACTOR * cutneigh) nexty++;

    nextz = (int) (cutneigh * bininvz);
    if(nextz * binsizez < FACTOR * cutneigh) nextz++;

    if (stencil) {
        free(stencil);
    }

    stencil = (int*) malloc(
            (2 * nextz + 1) * (2 * nexty + 1) * (2 * nextx + 1) * sizeof(int));

    nstencil = 0;
    int kstart = -nextz;

    for(int k = kstart; k <= nextz; k++) {
        for(int j = -nexty; j <= nexty; j++) {
            for(int i = -nextx; i <= nextx; i++) {
                if(bindist(i, j, k) < cutneighsq) {
                    stencil[nstencil++] =
                        k * mbiny * mbinx + j * mbinx + i;
                }
            }
        }
    }

    mbins = mbinx * mbiny * mbinz;

    if (bincount) {
        free(bincount);
    }
    bincount = (int*) malloc(mbins * sizeof(int));

    if (bins) {
        free(bins);
    }
    bins = (int*) malloc(mbins * atoms_per_bin * sizeof(int));
}

void buildNeighbor(Atom *atom, Neighbor *neighbor)
{
    int nall = atom->Nlocal + atom->Nghost;

    /* extend atom arrays if necessary */
    if(nall > nmax) {
        nmax = nall;
        if(neighbor->numneigh) hipHostFree(neighbor->numneigh);
        if(neighbor->neighbors) hipHostFree(neighbor->neighbors);
        checkCUDAError( "buildNeighbor numneigh", hipHostMalloc((void**)&(neighbor->numneigh), nmax * sizeof(int)) );
        checkCUDAError( "buildNeighbor neighbors", hipHostMalloc((void**)&(neighbor->neighbors), nmax * neighbor->maxneighs * sizeof(int)) );
        // neighbor->numneigh = (int*) malloc(nmax * sizeof(int));
        // neighbor->neighbors = (int*) malloc(nmax * neighbor->maxneighs * sizeof(int*));
    }

    /* bin local & ghost atoms */
    binatoms(atom);
    int resize = 1;

    /* loop over each atom, storing neighbors */
    while(resize) {
        int new_maxneighs = neighbor->maxneighs;
        resize = 0;

        for(int i = 0; i < atom->Nlocal; i++) {
            int* neighptr = &(neighbor->neighbors[i]);
            int n = 0;
            MD_FLOAT xtmp = atom_x(i);
            MD_FLOAT ytmp = atom_y(i);
            MD_FLOAT ztmp = atom_z(i);
            int ibin = coord2bin(xtmp, ytmp, ztmp);
            #ifdef EXPLICIT_TYPES
            int type_i = atom->type[i];
            #endif
            for(int k = 0; k < nstencil; k++) {
                int jbin = ibin + stencil[k];
                int* loc_bin = &bins[jbin * atoms_per_bin];

                for(int m = 0; m < bincount[jbin]; m++) {
                    int j = loc_bin[m];

                    if ( j == i ){
                        continue;
                    }

                    MD_FLOAT delx = xtmp - atom_x(j);
                    MD_FLOAT dely = ytmp - atom_y(j);
                    MD_FLOAT delz = ztmp - atom_z(j);
                    MD_FLOAT rsq = delx * delx + dely * dely + delz * delz;

                    #ifdef EXPLICIT_TYPES
                    int type_j = atom->type[j];
                    const MD_FLOAT cutoff = atom->cutneighsq[type_i * atom->ntypes + type_j];
                    #else
                    const MD_FLOAT cutoff = cutneighsq;
                    #endif

                    if( rsq <= cutoff ) {
                        int idx = atom->Nlocal * n;
                        neighptr[idx] = j;
                        n += 1;
                    }
                }
            }

            neighbor->numneigh[i] = n;

            if(n >= neighbor->maxneighs) {
                resize = 1;

                if(n >= new_maxneighs) {
                    new_maxneighs = n;
                }
            }
        }

        if(resize) {
            printf("RESIZE %d\n", neighbor->maxneighs);
            neighbor->maxneighs = new_maxneighs * 1.2;
            free(neighbor->neighbors);
            neighbor->neighbors = (int*) malloc(atom->Nmax * neighbor->maxneighs * sizeof(int));
        }
    }
}

/* internal subroutines */
MD_FLOAT bindist(int i, int j, int k)
{
    MD_FLOAT delx, dely, delz;

    if(i > 0) {
        delx = (i - 1) * binsizex;
    } else if(i == 0) {
        delx = 0.0;
    } else {
        delx = (i + 1) * binsizex;
    }

    if(j > 0) {
        dely = (j - 1) * binsizey;
    } else if(j == 0) {
        dely = 0.0;
    } else {
        dely = (j + 1) * binsizey;
    }

    if(k > 0) {
        delz = (k - 1) * binsizez;
    } else if(k == 0) {
        delz = 0.0;
    } else {
        delz = (k + 1) * binsizez;
    }

    return (delx * delx + dely * dely + delz * delz);
}

int coord2bin(MD_FLOAT xin, MD_FLOAT yin, MD_FLOAT zin)
{
    int ix, iy, iz;

    if(xin >= xprd) {
        ix = (int)((xin - xprd) * bininvx) + nbinx - mbinxlo;
    } else if(xin >= 0.0) {
        ix = (int)(xin * bininvx) - mbinxlo;
    } else {
        ix = (int)(xin * bininvx) - mbinxlo - 1;
    }

    if(yin >= yprd) {
        iy = (int)((yin - yprd) * bininvy) + nbiny - mbinylo;
    } else if(yin >= 0.0) {
        iy = (int)(yin * bininvy) - mbinylo;
    } else {
        iy = (int)(yin * bininvy) - mbinylo - 1;
    }

    if(zin >= zprd) {
        iz = (int)((zin - zprd) * bininvz) + nbinz - mbinzlo;
    } else if(zin >= 0.0) {
        iz = (int)(zin * bininvz) - mbinzlo;
    } else {
        iz = (int)(zin * bininvz) - mbinzlo - 1;
    }

    return (iz * mbiny * mbinx + iy * mbinx + ix + 1);
}

void binatoms(Atom *atom)
{
    int nall = atom->Nlocal + atom->Nghost;
    int resize = 1;

    while(resize > 0) {
        resize = 0;

        for(int i = 0; i < mbins; i++) {
            bincount[i] = 0;
        }

        for(int i = 0; i < nall; i++) {
            MD_FLOAT x = atom_x(i);
            MD_FLOAT y = atom_y(i);
            MD_FLOAT z = atom_z(i);
            int ibin = coord2bin(x, y, z);

            if(bincount[ibin] < atoms_per_bin) {
                int ac = bincount[ibin]++;
                bins[ibin * atoms_per_bin + ac] = i;
            } else {
                resize = 1;
            }
        }

        if(resize) {
            free(bins);
            atoms_per_bin *= 2;
            bins = (int*) malloc(mbins * atoms_per_bin * sizeof(int));
        }
    }
}

void sortAtom(Atom* atom) {
    binatoms(atom);
    int Nmax = atom->Nmax;
    int* binpos = bincount;

    for(int i=1; i<mbins; i++) {
        binpos[i] += binpos[i-1];
    }

#ifdef AOS
    double* new_x = (double*) malloc(Nmax * sizeof(MD_FLOAT) * 3);

    double* new_vx = (double*) malloc(Nmax * sizeof(MD_FLOAT) * 3);
#else
    double* new_x = (double*) malloc(Nmax * sizeof(MD_FLOAT));
    double* new_y = (double*) malloc(Nmax * sizeof(MD_FLOAT));
    double* new_z = (double*) malloc(Nmax * sizeof(MD_FLOAT));

    double* new_vx = (double*) malloc(Nmax * sizeof(MD_FLOAT));
    double* new_vy = (double*) malloc(Nmax * sizeof(MD_FLOAT));
    double* new_vz = (double*) malloc(Nmax * sizeof(MD_FLOAT));
#endif

    double* old_x = atom->x; double* old_y = atom->y; double* old_z = atom->z;
    double* old_vx = atom->vx; double* old_vy = atom->vy; double* old_vz = atom->vz;

    for(int mybin = 0; mybin<mbins; mybin++) {
        int start = mybin>0?binpos[mybin-1]:0;
        int count = binpos[mybin] - start;
        for(int k=0; k<count; k++) {
            int new_i = start + k;
            int old_i = bins[mybin * atoms_per_bin + k];
#ifdef AOS
            new_x[new_i * 3 + 0] = old_x[old_i * 3 + 0];
            new_x[new_i * 3 + 1] = old_x[old_i * 3 + 1];
            new_x[new_i * 3 + 2] = old_x[old_i * 3 + 2];

            new_vx[new_i * 3 + 0] = old_vx[old_i * 3 + 0];
            new_vx[new_i * 3 + 1] = old_vy[old_i * 3 + 1];
            new_vx[new_i * 3 + 2] = old_vz[old_i * 3 + 2];
#else
            new_x[new_i] = old_x[old_i];
            new_y[new_i] = old_y[old_i];
            new_z[new_i] = old_z[old_i];

            new_vx[new_i] = old_vx[old_i];
            new_vy[new_i] = old_vy[old_i];
            new_vz[new_i] = old_vz[old_i];
#endif

        }
    }

    free(atom->x);
    atom->x = new_x;

    free(atom->vx);
    atom->vx = new_vx;
#ifndef AOS
    free(atom->y);
    free(atom->z);
    atom->y = new_y; atom->z = new_z;

    free(atom->vy); free(atom->vz);
    atom->vy = new_vy; atom->vz = new_vz;
#endif
}

void buildNeighbor_cuda(Atom *atom, Neighbor *neighbor, Atom *c_atom, Neighbor *c_neighbor, const int num_threads_per_block)
{
    int nall = atom->Nlocal + atom->Nghost;

    c_atom->Natoms = atom->Natoms;
    c_atom->Nlocal = atom->Nlocal;
    c_atom->Nghost = atom->Nghost;
    c_atom->Nmax = atom->Nmax;
    c_atom->ntypes = atom->ntypes;

    c_neighbor->maxneighs = neighbor->maxneighs;

    /* extend c_neighbor arrays if necessary */
    if(nall > nmax) {
        nmax = nall;
        if(c_neighbor->numneigh) hipFree(c_neighbor->numneigh);
        if(c_neighbor->neighbors) hipFree(c_neighbor->neighbors);
        checkCUDAError( "buildNeighbor c_numneigh malloc", hipMalloc((void**)&(c_neighbor->numneigh), nmax * sizeof(int)) );
        checkCUDAError( "buildNeighbor c_neighbors malloc", hipMalloc((void**)&(c_neighbor->neighbors), nmax * c_neighbor->maxneighs * sizeof(int)) );
    }

    /* bin local & ghost atoms */
    binatoms(atom);
    int resize = 1;

    hipProfilerStart();

    checkCUDAError( "buildNeighbor c_atom->x memcpy", hipMemcpy(c_atom->x, atom->x, sizeof(MD_FLOAT) * atom->Nmax * 3, hipMemcpyHostToDevice) );

    /* upload stencil */
    int* c_stencil;
    // TODO move this to be done once at the start
    checkCUDAError( "buildNeighbor c_n_stencil malloc", hipMalloc((void**)&c_stencil, nstencil * sizeof(int)) );
    checkCUDAError( "buildNeighbor c_n_stencil memcpy", hipMemcpy(c_stencil, stencil, nstencil * sizeof(int), hipMemcpyHostToDevice ));

    int *c_bincount;
    checkCUDAError( "buildNeighbor c_bincount malloc", hipMalloc((void**)&c_bincount, mbins * sizeof(int)) );
    checkCUDAError( "buildNeighbor c_bincount memcpy", hipMemcpy(c_bincount, bincount, mbins * sizeof(int), hipMemcpyHostToDevice) );

    int *c_bins;
    checkCUDAError( "buidlNeighbor c_bins malloc", hipMalloc((void**)&c_bins, mbins * atoms_per_bin * sizeof(int)) );
    checkCUDAError( "buildNeighbor c_bins memcpy", hipMemcpy(c_bins, bins, mbins * atoms_per_bin * sizeof(int), hipMemcpyHostToDevice ) );

    Neighbor_params np{
        .xprd = xprd,
        .yprd = yprd,
        .zprd = zprd,
        .bininvx = bininvx,
        .bininvy = bininvy,
        .bininvz = bininvz,
        .mbinxlo = mbinxlo,
        .mbinylo = mbinylo,
        .mbinzlo = mbinzlo,
        .nbinx = nbinx,
        .nbiny = nbiny,
        .nbinz = nbinz,
        .mbinx = mbinx,
        .mbiny = mbiny,
        .mbinz = mbinz
    };

    int* c_new_maxneighs;
    checkCUDAError("c_new_maxneighs malloc", hipMalloc((void**)&c_new_maxneighs, sizeof(int) ));

    /* loop over each atom, storing neighbors */
    while(resize) {
        resize = 0;

        checkCUDAError("c_new_maxneighs memset", hipMemset(c_new_maxneighs, 0, sizeof(int) ));

        // TODO call compute_neigborhood kernel here
        const int num_blocks = ceil((float)atom->Nlocal / (float)num_threads_per_block);
        /*compute_neighborhood(Atom a, Neighbor neigh, Neighbor_params np, int nstencil, int* stencil,
                                     int* bins, int atoms_per_bin, int *bincount, int *new_maxneighs)
         * */
        compute_neighborhood<<<num_blocks, num_threads_per_block>>>(*c_atom, *c_neighbor,
                                                                    np, nstencil, c_stencil,
                                                                    c_bins, atoms_per_bin, c_bincount,
                                                                    c_new_maxneighs,
								                                    cutneighsq);

	checkCUDAError( "PeekAtLastError ComputeNeighbor", hipPeekAtLastError() );
	checkCUDAError( "DeviceSync ComputeNeighbor", hipDeviceSynchronize() );

        // TODO copy the value of c_new_maxneighs back to host and check if it has been modified
        int new_maxneighs;
        checkCUDAError("c_new_maxneighs memcpy back", hipMemcpy(&new_maxneighs, c_new_maxneighs, sizeof(int), hipMemcpyDeviceToHost));
        if (new_maxneighs > c_neighbor->maxneighs){
            resize = 1;
        }

        if(resize) {
            printf("RESIZE %d\n", c_neighbor->maxneighs);
            c_neighbor->maxneighs = new_maxneighs * 1.2;
            printf("NEW SIZE %d\n", c_neighbor->maxneighs);
            hipFree(c_neighbor->neighbors);
            checkCUDAError("c_neighbor->neighbors resize malloc",
                           hipMalloc((void**)(&c_neighbor->neighbors),
                                      c_atom->Nmax * c_neighbor->maxneighs * sizeof(int)));
        }

    }
    neighbor->maxneighs = c_neighbor->maxneighs;

    hipProfilerStop();

    hipFree(c_new_maxneighs);
    hipFree(c_stencil);
    hipFree(c_bincount);
    hipFree(c_bins);
}
}
