#include "hip/hip_runtime.h"
/*
 * =======================================================================================
 *
 *   Author:   Jan Eitzinger (je), jan.eitzinger@fau.de
 *   Copyright (c) 2021 RRZE, University Erlangen-Nuremberg
 *
 *   This file is part of MD-Bench.
 *
 *   MD-Bench is free software: you can redistribute it and/or modify it
 *   under the terms of the GNU Lesser General Public License as published
 *   by the Free Software Foundation, either version 3 of the License, or
 *   (at your option) any later version.
 *
 *   MD-Bench is distributed in the hope that it will be useful, but WITHOUT ANY
 *   WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A
 *   PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 *   details.
 *
 *   You should have received a copy of the GNU Lesser General Public License along
 *   with MD-Bench.  If not, see <https://www.gnu.org/licenses/>.
 * =======================================================================================
 */
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <stddef.h>
#include <hip/hip_runtime.h>
#include <>

extern "C" {
    #include <likwid-marker.h>

    #include <timing.h>
    #include <neighbor.h>
    #include <parameter.h>
    #include <atom.h>
}

// cuda kernel
__global__ void calc_force(
    Atom a,
    MD_FLOAT xtmp, MD_FLOAT ytmp, MD_FLOAT ztmp,
    MD_FLOAT *fix, MD_FLOAT *fiy, MD_FLOAT *fiz,
    int i, int numneighs, int *neighs) {

    // Calculate idx k from thread information
    const long long k = blockIdx.x * blockDim.x + threadIdx.x;
    if( k >= numneighs ) {
        return;
    }

    Atom *atom = &a;

    int j = neighs[k];
    MD_FLOAT delx = xtmp - atom_x(j);
    MD_FLOAT dely = ytmp - atom_y(j);
    MD_FLOAT delz = ztmp - atom_z(j);
    MD_FLOAT rsq = delx * delx + dely * dely + delz * delz;

    const int type_i = atom->type[i];
    const int type_j = atom->type[j];
    const int type_ij = type_i * atom->ntypes + type_j;
    const MD_FLOAT cutforcesq = atom->cutforcesq[type_ij];
    const MD_FLOAT sigma6 = atom->sigma6[type_ij];
    const MD_FLOAT epsilon = atom->epsilon[type_ij];

    if(rsq < cutforcesq) {
        MD_FLOAT sr2 = 1.0 / rsq;
        MD_FLOAT sr6 = sr2 * sr2 * sr2 * sigma6;
        MD_FLOAT force = 48.0 * sr6 * (sr6 - 0.5) * sr2 * epsilon;
        fix[j] = delx * force;
        fiy[j] = dely * force;
        fiz[j] = delz * force;
    }
}

extern "C" {

double computeForce(
        Parameter *param,
        Atom *atom,
        Neighbor *neighbor
        )
{
    int Nlocal = atom->Nlocal;
    int* neighs;
    MD_FLOAT* fx = atom->fx;
    MD_FLOAT* fy = atom->fy;
    MD_FLOAT* fz = atom->fz;
#ifndef EXPLICIT_TYPES
    MD_FLOAT cutforcesq = param->cutforce * param->cutforce;
#endif

    for(int i = 0; i < Nlocal; i++) {
        fx[i] = 0.0;
        fy[i] = 0.0;
        fz[i] = 0.0;
    }

    double S = getTimeStamp();
    LIKWID_MARKER_START("force");

#pragma omp parallel for
    for(int i = 0; i < Nlocal; i++) {
        neighs = &neighbor->neighbors[i * neighbor->maxneighs];
        int numneighs = neighbor->numneigh[i];
        MD_FLOAT xtmp = atom_x(i);
        MD_FLOAT ytmp = atom_y(i);
        MD_FLOAT ztmp = atom_z(i);

#ifdef EXPLICIT_TYPES
        const int type_i = atom->type[i];
#endif

        Atom c_atom;
        memcpy(&c_atom, atom, sizeof(Atom));

        hipMalloc((void**)&(&c_atom)->x, sizeof(MD_FLOAT) * atom->Nmax * 3);
        hipMemcpy(c_atom.x, atom->x, sizeof(MD_FLOAT) * atom->Nmax * 3, hipMemcpyHostToDevice);

        hipMalloc((void**)&(&c_atom)->y, sizeof(MD_FLOAT) * atom->Nmax * 3);
        hipMemcpy(c_atom.y, atom->y, sizeof(MD_FLOAT) * atom->Nmax * 3, hipMemcpyHostToDevice);

        hipMalloc((void**)&(&c_atom)->z, sizeof(MD_FLOAT) * atom->Nmax * 3);
        hipMemcpy(c_atom.z, atom->z, sizeof(MD_FLOAT) * atom->Nmax * 3, hipMemcpyHostToDevice);

        hipMalloc((void**)&(&c_atom)->type, sizeof(int) * atom->Nmax);
        hipMemcpy(c_atom.type, atom->type, sizeof(int) * atom->Nmax, hipMemcpyHostToDevice);

        hipMalloc((void**)&(&c_atom)->epsilon, sizeof(MD_FLOAT) * atom->ntypes * atom->ntypes);
        hipMemcpy(c_atom.epsilon, atom->epsilon, sizeof(MD_FLOAT) * atom->ntypes * atom->ntypes, hipMemcpyHostToDevice);

        hipMalloc((void**)&(&c_atom)->sigma6, sizeof(MD_FLOAT) * atom->ntypes * atom->ntypes);
        hipMemcpy(c_atom.sigma6, atom->sigma6, sizeof(MD_FLOAT) * atom->ntypes * atom->ntypes, hipMemcpyHostToDevice);

        hipMalloc((void**)&(&c_atom)->cutforcesq, sizeof(MD_FLOAT) * atom->ntypes * atom->ntypes);
        hipMemcpy(c_atom.cutforcesq, atom->cutforcesq, sizeof(MD_FLOAT) * atom->ntypes * atom->ntypes, hipMemcpyHostToDevice);

        int *c_neighs;
        hipMalloc((void**)&c_neighs, sizeof(int) * numneighs);
        hipMemcpy(c_neighs, neighs, sizeof(int) * numneighs, hipMemcpyHostToDevice);

        MD_FLOAT *c_fix, *c_fiy, *c_fiz;
        hipMalloc((void**)&c_fix, sizeof(MD_FLOAT) * numneighs);
        hipMalloc((void**)&c_fiy, sizeof(MD_FLOAT) * numneighs);
        hipMalloc((void**)&c_fiz, sizeof(MD_FLOAT) * numneighs);

        const int num_blocks = 64;
        const int num_threads_per_block = ceil((float)numneighs / (float)num_blocks);
        // printf("numneighs: %d => num-blocks: %d, num_threads_per_block => %d\r\n", numneighs, num_blocks, num_threads_per_block);

        // launch cuda kernel
        calc_force <<< num_blocks, num_threads_per_block >>> (c_atom, xtmp, ytmp, ztmp, c_fix, c_fiy, c_fiz, i, numneighs, c_neighs);
        hipDeviceSynchronize();

        // sum result
        MD_FLOAT *d_fix, *d_fiy, *d_fiz;
        d_fix = (MD_FLOAT*)malloc(sizeof(MD_FLOAT) * numneighs);
        d_fiy = (MD_FLOAT*)malloc(sizeof(MD_FLOAT) * numneighs);
        d_fiz = (MD_FLOAT*)malloc(sizeof(MD_FLOAT) * numneighs);
        hipMemcpy((void**)&d_fix, c_fix, sizeof(MD_FLOAT) * numneighs, hipMemcpyDeviceToHost);
        hipMemcpy((void**)&d_fiy, c_fiy, sizeof(MD_FLOAT) * numneighs, hipMemcpyDeviceToHost);
        hipMemcpy((void**)&d_fiz, c_fiz, sizeof(MD_FLOAT) * numneighs, hipMemcpyDeviceToHost);

        for(int k = 0; k < numneighs; k++) {
            fx[i] += d_fix[k];
            fy[i] += d_fiy[k];
            fz[i] += d_fiz[k];
        }

        hipFree(c_fix); hipFree(c_fiy); hipFree(c_fiz); hipFree(c_neighs);
        hipFree(c_atom.x); hipFree(c_atom.y); hipFree(c_atom.z); hipFree(c_atom.type);
        hipFree(c_atom.epsilon); hipFree(c_atom.sigma6); hipFree(c_atom.cutforcesq);

        free(d_fix); free(d_fiy); free(d_fiz);
    }

    LIKWID_MARKER_STOP("force");
    double E = getTimeStamp();

    return E-S;
}
}