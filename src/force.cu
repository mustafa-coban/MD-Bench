#include "hip/hip_runtime.h"
/*
 * =======================================================================================
 *
 *   Author:   Jan Eitzinger (je), jan.eitzinger@fau.de
 *   Copyright (c) 2021 RRZE, University Erlangen-Nuremberg
 *
 *   This file is part of MD-Bench.
 *
 *   MD-Bench is free software: you can redistribute it and/or modify it
 *   under the terms of the GNU Lesser General Public License as published
 *   by the Free Software Foundation, either version 3 of the License, or
 *   (at your option) any later version.
 *
 *   MD-Bench is distributed in the hope that it will be useful, but WITHOUT ANY
 *   WARRANTY; without even the implied warranty of MERCHANTABILITY or FITNESS FOR A
 *   PARTICULAR PURPOSE.  See the GNU Lesser General Public License for more
 *   details.
 *
 *   You should have received a copy of the GNU Lesser General Public License along
 *   with MD-Bench.  If not, see <https://www.gnu.org/licenses/>.
 * =======================================================================================
 */
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <stddef.h>
#include <hip/hip_runtime.h>
#include <>

extern "C" {
    #include <likwid-marker.h>

    #include <timing.h>
    #include <neighbor.h>
    #include <parameter.h>
    #include <atom.h>
}

void checkError(const char *msg, hipError_t err)
{
    if (err != hipSuccess)
    {
        //print a human readable error message
        printf("[CUDA ERROR %s]: %s\r\n", msg, hipGetErrorString(err));
        exit(-1);
    }
}

// cuda kernel
__global__ void calc_force(
    Atom a,
    MD_FLOAT cutforcesq, MD_FLOAT sigma6, MD_FLOAT epsilon,
    int Nlocal, int neigh_maxneighs, int *neigh_neighbors, int *neigh_numneigh) {

    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    if( i >= Nlocal ) {
        return;
    }

    Atom *atom = &a;

    int *neighs = &neigh_neighbors[i * neigh_maxneighs];
    int numneighs = neigh_numneigh[i];

    MD_FLOAT xtmp = atom_x(i);
    MD_FLOAT ytmp = atom_y(i);
    MD_FLOAT ztmp = atom_z(i);

    MD_FLOAT *fx = atom->fx;
    MD_FLOAT *fy = atom->fy;
    MD_FLOAT *fz = atom->fz;

    MD_FLOAT fix = 0;
    MD_FLOAT fiy = 0;
    MD_FLOAT fiz = 0;

    for(int k = 0; k < numneighs; k++) {
        int j = neighs[k];
        MD_FLOAT delx = xtmp - atom_x(j);
        MD_FLOAT dely = ytmp - atom_y(j);
        MD_FLOAT delz = ztmp - atom_z(j);
        MD_FLOAT rsq = delx * delx + dely * dely + delz * delz;

#ifdef EXPLICIT_TYPES
        const int type_j = atom->type[j];
        const int type_ij = type_i * atom->ntypes + type_j;
        const MD_FLOAT cutforcesq = atom->cutforcesq[type_ij];
        const MD_FLOAT sigma6 = atom->sigma6[type_ij];
        const MD_FLOAT epsilon = atom->epsilon[type_ij];
#endif

        if(rsq < cutforcesq) {
            MD_FLOAT sr2 = 1.0 / rsq;
            MD_FLOAT sr6 = sr2 * sr2 * sr2 * sigma6;
            MD_FLOAT force = 48.0 * sr6 * (sr6 - 0.5) * sr2 * epsilon;
            fix += delx * force;
            fiy += dely * force;
            fiz += delz * force;
        }
    }

    fx[i] += fix;
    fy[i] += fiy;
    fz[i] += fiz;
}

extern "C" {

double computeForce(
        Parameter *param,
        Atom *atom,
        Neighbor *neighbor
        )
{
    int Nlocal = atom->Nlocal;
    MD_FLOAT* fx = atom->fx;
    MD_FLOAT* fy = atom->fy;
    MD_FLOAT* fz = atom->fz;
#ifndef EXPLICIT_TYPES
    MD_FLOAT cutforcesq = param->cutforce * param->cutforce;
    MD_FLOAT sigma6 = param->sigma6;
    MD_FLOAT epsilon = param->epsilon;
#endif

    for(int i = 0; i < Nlocal; i++) {
        fx[i] = 0.0;
        fy[i] = 0.0;
        fz[i] = 0.0;
    }

    const char *num_threads_env = getenv("NUM_THREADS");
    const int num_threads = atoi(num_threads_env);

    Atom c_atom;
    c_atom.Natoms = atom->Natoms;
    c_atom.Nlocal = atom->Nlocal;
    c_atom.Nghost = atom->Nghost;
    c_atom.Nmax = atom->Nmax;
    c_atom.ntypes = atom->ntypes;

    /*
    int nDevices;
    hipGetDeviceCount(&nDevices);
    for(int i = 0; i < nDevices; ++i) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("DEVICE NAME: %s\r\n", prop.name);
    }
    */

    // HINT: Run with cuda-memcheck ./MDBench-NVCC in case of error
    // HINT: Only works for data layout = AOS!!!

    checkError( "c_atom.x malloc", hipMalloc((void**)&(c_atom.x), sizeof(MD_FLOAT) * atom->Nmax * 3) );
    checkError( "c_atom.x memcpy", hipMemcpy(c_atom.x, atom->x, sizeof(MD_FLOAT) * atom->Nmax * 3, hipMemcpyHostToDevice) );

    checkError( "c_atom.fx malloc", hipMalloc((void**)&(c_atom.fx), sizeof(MD_FLOAT) * Nlocal) );
    checkError( "c_atom.fx memcpy", hipMemcpy(c_atom.fx, fx, sizeof(MD_FLOAT) * Nlocal, hipMemcpyHostToDevice) );

    checkError( "c_atom.fy malloc", hipMalloc((void**)&(c_atom.fy), sizeof(MD_FLOAT) * Nlocal) );
    checkError( "c_atom.fy memcpy", hipMemcpy(c_atom.fy, fy, sizeof(MD_FLOAT) * Nlocal, hipMemcpyHostToDevice) );

    checkError( "c_atom.fz malloc", hipMalloc((void**)&(c_atom.fz), sizeof(MD_FLOAT) * Nlocal) );
    checkError( "c_atom.fz memcpy", hipMemcpy(c_atom.fz, fz, sizeof(MD_FLOAT) * Nlocal, hipMemcpyHostToDevice) );

    checkError( "c_atom.type malloc", hipMalloc((void**)&(c_atom.type), sizeof(int) * atom->Nmax) );
    checkError( "c_atom.type memcpy", hipMemcpy(c_atom.type, atom->type, sizeof(int) * atom->Nmax, hipMemcpyHostToDevice) );

    checkError( "c_atom.epsilon malloc", hipMalloc((void**)&(c_atom.epsilon), sizeof(MD_FLOAT) * atom->ntypes * atom->ntypes) );
    checkError( "c_atom.epsilon memcpy", hipMemcpy(c_atom.epsilon, atom->epsilon, sizeof(MD_FLOAT) * atom->ntypes * atom->ntypes, hipMemcpyHostToDevice) );

    checkError( "c_atom.sigma6 malloc", hipMalloc((void**)&(c_atom.sigma6), sizeof(MD_FLOAT) * atom->ntypes * atom->ntypes) );
    checkError( "c_atom.sigma6 memcpy", hipMemcpy(c_atom.sigma6, atom->sigma6, sizeof(MD_FLOAT) * atom->ntypes * atom->ntypes, hipMemcpyHostToDevice) );

    checkError( "c_atom.cutforcesq malloc", hipMalloc((void**)&(c_atom.cutforcesq), sizeof(MD_FLOAT) * atom->ntypes * atom->ntypes) );
    checkError( "c_atom.cutforcesq memcpy", hipMemcpy(c_atom.cutforcesq, atom->cutforcesq, sizeof(MD_FLOAT) * atom->ntypes * atom->ntypes, hipMemcpyHostToDevice) );


    // double start_memory_bandwidth = getTimeStamp();

    int *c_neighs;
    checkError( "c_neighs malloc", hipMalloc((void**)&c_neighs, sizeof(int) * Nlocal * neighbor->maxneighs) );
    checkError( "c_neighs memcpy", hipMemcpy(c_neighs, neighbor->neighbors, sizeof(int) * Nlocal * neighbor->maxneighs, hipMemcpyHostToDevice) );

    /*
    double end_memory_bandwidth = getTimeStamp();
    double memory_bandwith_time = (end_memory_bandwidth - start_memory_bandwidth);
    const unsigned long bytes =  sizeof(int) * Nlocal * neighbor->maxneighs;
    const double gb_per_second = ((double)bytes / memory_bandwith_time) / 1024.0 / 1024.0 / 1024.0;
    printf("Data transfer of %lu bytes took %fs => %f GB/s\r\n", bytes, memory_bandwith_time, gb_per_second);
    */

    int *c_neigh_numneigh;
    checkError( "c_neigh_numneigh malloc", hipMalloc((void**)&c_neigh_numneigh, sizeof(int) * Nlocal) );
    checkError( "c_neigh_numneigh memcpy", hipMemcpy(c_neigh_numneigh, neighbor->numneigh, sizeof(int) * Nlocal, hipMemcpyHostToDevice) );

    const int num_threads_per_block = num_threads; // this should be multiple of 32 as operations are performed at the level of warps
    const int num_blocks = ceil((float)Nlocal / (float)num_threads_per_block);
    // printf("Distribution size: %d\r\n%d Blocks with each %d threads\r\n", Nlocal, num_blocks, num_threads_per_block);

    double S = getTimeStamp();
    LIKWID_MARKER_START("force");

    calc_force <<< num_blocks, num_threads_per_block >>> (c_atom, cutforcesq, sigma6, epsilon, Nlocal, neighbor->maxneighs, c_neighs, c_neigh_numneigh);

    checkError( "PeekAtLastError", hipPeekAtLastError() );
    checkError( "DeviceSync", hipDeviceSynchronize() );

    // copy results in c_atom.fx/fy/fz to atom->fx/fy/fz
    hipMemcpy(atom->fx, c_atom.fx, sizeof(MD_FLOAT) * Nlocal, hipMemcpyDeviceToHost);
    hipMemcpy(atom->fy, c_atom.fy, sizeof(MD_FLOAT) * Nlocal, hipMemcpyDeviceToHost);
    hipMemcpy(atom->fz, c_atom.fz, sizeof(MD_FLOAT) * Nlocal, hipMemcpyDeviceToHost);

    hipFree(c_atom.x);
    hipFree(c_atom.fx); hipFree(c_atom.fy); hipFree(c_atom.fz);
    hipFree(c_atom.type);
    hipFree(c_atom.epsilon);
    hipFree(c_atom.sigma6);
    hipFree(c_atom.cutforcesq);

    hipFree(c_neighs); hipFree(c_neigh_numneigh);

    LIKWID_MARKER_STOP("force");
    double E = getTimeStamp();

    return E-S;
}
}